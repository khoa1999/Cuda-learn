
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 16
#define BLOCK_SIZE 4
__global__ void transpose(int *input,int *output){
	__shared__ int sharedMemory[BLOCK_SIZE][BLOCK_SIZE + 1];
	//global index
	int indexX = threadIdx.x + blockIdx.x*blockDim.x;
	int indexY = threadIdx.y + blockIdx.y*blockDim.y;
	//transposed index
	int tindexX = threadIdx.x + blockIdx.y*blockDim.x;
	int tindexY = threadIdx.y + blockIdx.x*blockDim.y;
	//local index
	int localIndexX = threadIdx.x;
	int localIndexY = threadIdx.y;
	int index = indexX*N + indexY;
	int transposedIndex = tindexY*N + tindexX;
	sharedMemory[localIndexX][localIndexY] = input[index];
	__syncthreads();
	output[transposedIndex] = sharedMemory[localIndexY][localIndexX];
}	
void fill_data(int *data){
	for(int idx=0;idx < N*N;idx++)
		data[idx] = idx;
}
void print_matrix(int *data,int n){
	for(int i = 0;i < n;i++){
		for(int j = 0;j < n;j++){
			printf("%4d ",data[i*n + j]);
		}
		printf("\n");
	}
}
int main(void){
	int *a,*b;
	int *d_a,*d_b;
	int size = N*N*sizeof(int);
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	fill_data(a);
	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE,1);
	dim3 gridSize(N/BLOCK_SIZE,N/BLOCK_SIZE,1);
	transpose<<<blockSize,gridSize>>>(d_a,d_b);
	hipMemcpy(b,d_b,size,hipMemcpyDeviceToHost);
	printf("Original:\n");
	print_matrix(a,N);
	printf("Transposed:\n");
	print_matrix(b,N);
	free(a);
	free(b);
	hipFree(d_a);
	hipFree(d_b);
}

